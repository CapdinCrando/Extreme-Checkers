#include "hip/hip_runtime.h"
#include "gpuutility.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <hipblas.h>
#include <stdio.h>

// To get rid of intellisense warnings
#ifdef CUDA_EDIT
#define CUDA_KERNEL(...)
#define __syncthreads()
template<class T1, class T2>
__device__ void atomicMax(T1 x, T2 y);
template<class T1, class T2>
__device__ void atomicMin(T1 x, T2 y);
template<class T, class T2>
__device__ T atomicAdd(T* x, T2 y);
template<class T, class T2>
__device__ T atomicCAS(T* x, T2 y, T2 z);
#else
#define CUDA_KERNEL(...) <<<__VA_ARGS__>>>
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
__device__ inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
	  printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
   }
}

typedef int result_gpu_t;
#define IS_ROOT_THREAD threadIdx.x == 0
#define MOVE_BUFFER_SIZE SQUARE_COUNT

__device__ boardpos_t previousMultiJumpPosGPU = -1;

__constant__ const boardpos_t cornerListDev[SQUARE_COUNT][4] = {
	{-1, -1, 4, 5},{-1, -1, 5, 6},{-1, -1, 6, 7},{-1, -1, 7, -1},
	{-1, 0, -1, 8},{0, 1, 8, 9},{1, 2, 9, 10},{2, 3, 10, 11},
	{4, 5, 12, 13},{5, 6, 13, 14},{6, 7, 14, 15},{7, -1, 15, -1},
	{-1, 8, -1, 16},{8, 9, 16, 17},{9, 10, 17, 18},{10, 11, 18, 19},
	{12, 13, 20, 21},{13, 14, 21, 22},{14, 15, 22, 23},{15, -1, 23, -1},
	{-1, 16, -1, 24},{16, 17, 24, 25},{17, 18, 25, 26},{18, 19, 26, 27},
	{20, 21, 28, 29},{21, 22, 29, 30},{22, 23, 30, 31},{23, -1, 31, -1},
	{-1, 24, -1, -1},{24, 25, -1, -1},{25, 26, -1, -1},{26, 27, -1, -1}
};


__device__ void getBlackJumpsGPU(Move* jumpsOut, unsigned int& jumpCount, boardpos_t pos, boardstate_t* board, boardpos_t (&cornerTile)[SQUARE_COUNT][4])
{
	__shared__ Move jumps[MOVE_BUFFER_SIZE];
	unsigned int i = pos;

	unsigned int j = threadIdx.x;
	if(j < 4)
	{
		boardstate_t state = board[i];
		if(SQUARE_ISNOTEMPTY(state))
		{
			if(SQUARE_ISBLACK(state))
			{
				// Get move
				boardpos_t move = cornerTile[i][j];
				// Check if position is invalid
				if(move != BOARD_POS_INVALID)
				{
					// Check if space is empty
					boardstate_t moveState = board[move];
					if(SQUARE_ISNOTEMPTY(moveState))
					{
						if(!(SQUARE_ISBLACK(moveState)))
						{
							// Get jump
							boardpos_t jump = cornerTile[move][j];
							// Check if position is invalid
							if(jump != BOARD_POS_INVALID)
							{
								// Check if space is empty
								if(SQUARE_ISEMPTY(board[jump]))
								{
									// Add move to potential moves
									uint16_t jumpIndex = atomicAdd(&jumpCount, 1U);
									jumps[jumpIndex].oldPos = i;
									jumps[jumpIndex].newPos = jump;
									jumps[jumpIndex].jumpPos = move;
									// Check for multi
									jumps[jumpIndex].moveType = MOVE_JUMP;
									for(uint8_t k = 0; k < 4; k++)
									{
										boardpos_t moveMulti = cornerTile[jump][k];
										// Check if position is invalid
										if(moveMulti != BOARD_POS_INVALID)
										{
											if(moveMulti != move)
											{
												boardstate_t moveStateMulti = board[moveMulti];
												if(SQUARE_ISNOTEMPTY(moveStateMulti))
												{
													if(!(SQUARE_ISBLACK(moveStateMulti)))
													{
														boardpos_t jumpMulti = cornerTile[moveMulti][k];
														if(jumpMulti != BOARD_POS_INVALID)
														{
															boardstate_t jumpStateMulti = board[jumpMulti];
															if(SQUARE_ISEMPTY(jumpStateMulti))
															{
																jumps[jumpIndex].moveType = MOVE_JUMP_MULTI;
																break;
															}
														}
													}
												}
											}
										}
									}
								}
							}
						}
					}
				}
			}
		}
	}
	__syncthreads();
	if(j < jumpCount) jumpsOut[j] = jumps[j];
	__syncthreads();
}

__device__ void getRedJumpsGPU(Move* jumpsOut, unsigned int& jumpCount, boardpos_t pos, boardstate_t* board, boardpos_t (&cornerTile)[SQUARE_COUNT][4])
{
	__shared__ Move jumps[MOVE_BUFFER_SIZE];
	unsigned int i = pos;

	unsigned int j = threadIdx.x;
	if(j < 4)
	{
		boardstate_t state = board[i];
		if(SQUARE_ISNOTEMPTY(state))
		{
			if(!(SQUARE_ISBLACK(state)))
			{
				// Get move
				boardpos_t move = cornerTile[i][j];
				// Check if position is invalid
				if(move != BOARD_POS_INVALID)
				{
					// Check if space is empty
					boardstate_t moveState = board[move];
					if(SQUARE_ISNOTEMPTY(moveState))
					{
						if(SQUARE_ISBLACK(moveState))
						{
							// Get jump
							boardpos_t jump = cornerTile[move][j];
							// Check if position is invalid
							if(jump != BOARD_POS_INVALID)
							{
								// Check if space is empty
								if(SQUARE_ISEMPTY(board[jump]))
								{
									// Add move to potential moves
									uint16_t jumpIndex = atomicAdd(&jumpCount, 1U);
									jumps[jumpIndex].oldPos = i;
									jumps[jumpIndex].newPos = jump;
									jumps[jumpIndex].jumpPos = move;
									// Check for multi
									jumps[jumpIndex].moveType = MOVE_JUMP;
									for(uint8_t k = 0; k < 4; k++)
									{
										boardpos_t moveMulti = cornerTile[jump][k];
										// Check if position is invalid
										if(moveMulti != BOARD_POS_INVALID)
										{
											if(moveMulti != move)
											{
												boardstate_t moveStateMulti = board[moveMulti];
												if(SQUARE_ISNOTEMPTY(moveStateMulti))
												{
													if(SQUARE_ISBLACK(moveStateMulti))
													{
														boardpos_t jumpMulti = cornerTile[moveMulti][k];
														if(jumpMulti != BOARD_POS_INVALID)
														{
															boardstate_t jumpStateMulti = board[jumpMulti];
															if(SQUARE_ISEMPTY(jumpStateMulti))
															{
																jumps[jumpIndex].moveType = MOVE_JUMP_MULTI;
																break;
															}
														}
													}
												}
											}
										}
									}
								}
							}
						}
					}
				}
			}
		}
	}
	__syncthreads();
	if(j < jumpCount) jumpsOut[j] = jumps[j];
	__syncthreads();
}

__device__ void getBlackMovesGPU(Move* movesOut, unsigned int& moveCount, boardstate_t* board, boardpos_t (&cornerTile)[SQUARE_COUNT][4])
{
	__shared__ Move moves[MOVE_BUFFER_SIZE];
	__shared__ Move jumps[MOVE_BUFFER_SIZE];
	__shared__ unsigned int jumpCount;
	if(IS_ROOT_THREAD) jumpCount = 0;
	__syncthreads();

	unsigned int i = threadIdx.x;

	boardstate_t state = board[i];
	if(SQUARE_ISNOTEMPTY(state))
	{
		if(SQUARE_ISBLACK(state))
		{
			uint8_t cornerMin = 2;
			if(SQUARE_ISKING(state)) cornerMin = 0;
			for(uint8_t j = cornerMin; j < 4; j++)
			{
				// Get move
				boardpos_t move = cornerTile[i][j];
				// Check if position is invalid
				if(move != BOARD_POS_INVALID)
				{
					// Check if space is empty
					boardstate_t moveState = board[move];
					if(SQUARE_ISEMPTY(moveState))
					{
						// Add move to potential moves
						uint16_t moveIndex = atomicAdd(&moveCount, 1U);
						moves[moveIndex].oldPos = i;
						moves[moveIndex].newPos = move;
						moves[moveIndex].moveType = MOVE_MOVE;
					}
					else if(!(SQUARE_ISBLACK(moveState)))
					{
						// Get jump
						boardpos_t jump = cornerTile[move][j];
						// Check if position is invalid
						if(jump != BOARD_POS_INVALID)
						{
							// Check if space is empty
							if(SQUARE_ISEMPTY(board[jump]))
							{
								// Add move to potential moves
								uint16_t jumpIndex = atomicAdd(&jumpCount, 1U);
								jumps[jumpIndex].oldPos = i;
								jumps[jumpIndex].newPos = jump;
								jumps[jumpIndex].jumpPos = move;
								// Check for multi
								jumps[jumpIndex].moveType = MOVE_JUMP;
								for(uint8_t k = 0; k < 4; k++)
								{
									boardpos_t moveMulti = cornerTile[jump][k];
									// Check if position is invalid
									if(moveMulti != BOARD_POS_INVALID)
									{
										if(moveMulti != move)
										{
											boardstate_t moveStateMulti = board[moveMulti];
											if(SQUARE_ISNOTEMPTY(moveStateMulti))
											{
												if(!(SQUARE_ISBLACK(moveStateMulti)))
												{
													boardpos_t jumpMulti = cornerTile[moveMulti][k];
													if(jumpMulti != BOARD_POS_INVALID)
													{
														boardstate_t jumpStateMulti = board[jumpMulti];
														if(SQUARE_ISEMPTY(jumpStateMulti))
														{
															jumps[jumpIndex].moveType = MOVE_JUMP_MULTI;
															break;
														}
													}
												}
											}
										}
									}
								}
							}
						}
					}
				}
			}
		}
	}
	__syncthreads();

	if(jumpCount)
	{
		if(IS_ROOT_THREAD) moveCount = jumpCount;
		if(i < jumpCount) movesOut[i] = jumps[i];
	}
	else if(i < moveCount) movesOut[i] = moves[i];
	__syncthreads();
}

__device__ void getRedMovesGPU(Move* movesOut, unsigned int& moveCount, boardstate_t* board, boardpos_t (&cornerTile)[SQUARE_COUNT][4])
{
	__shared__ Move moves[MOVE_BUFFER_SIZE];
	__shared__ Move jumps[MOVE_BUFFER_SIZE];
	__shared__ unsigned int jumpCount;
	if(IS_ROOT_THREAD) jumpCount = 0;
	__syncthreads();

	unsigned int i = threadIdx.x;

	boardstate_t state = board[i];
	if(SQUARE_ISNOTEMPTY(state))
	{
		if(!(SQUARE_ISBLACK(state)))
		{
			uint8_t cornerMax = 2;
			if(SQUARE_ISKING(state)) cornerMax = 4;
			for(uint8_t j = 0; j < cornerMax; j++)
			{
				// Get move
				boardpos_t move = cornerTile[i][j];
				// Check if position is invalid
				if(move != BOARD_POS_INVALID)
				{
					// Check if space is empty
					boardstate_t moveState = board[move];
					if(SQUARE_ISEMPTY(moveState))
					{
						// Add move to potential moves
						uint16_t moveIndex = atomicAdd(&moveCount, 1U);
						moves[moveIndex].oldPos = i;
						moves[moveIndex].newPos = move;
						moves[moveIndex].moveType = MOVE_MOVE;
					}
					else if(SQUARE_ISBLACK(moveState))
					{
						// Get jump
						boardpos_t jump = cornerTile[move][j];
						// Check if position is invalid
						if(jump != BOARD_POS_INVALID)
						{
							// Check if space is empty
							if(SQUARE_ISEMPTY(board[jump]))
							{
								// Add move to potential moves
								uint16_t jumpIndex = atomicAdd(&jumpCount, 1U);
								jumps[jumpIndex].oldPos = i;
								jumps[jumpIndex].newPos = jump;
								jumps[jumpIndex].jumpPos = move;
								// Check for multi
								jumps[jumpIndex].moveType = MOVE_JUMP;
								for(uint8_t k = 0; k < 4; k++)
								{
									boardpos_t moveMulti = cornerTile[jump][k];
									// Check if position is invalid
									if(moveMulti != BOARD_POS_INVALID)
									{
										if(moveMulti != move)
										{
											boardstate_t moveStateMulti = board[moveMulti];
											if(SQUARE_ISNOTEMPTY(moveStateMulti))
											{
												if(SQUARE_ISBLACK(moveStateMulti))
												{
													boardpos_t jumpMulti = cornerTile[moveMulti][k];
													if(jumpMulti != BOARD_POS_INVALID)
													{
														boardstate_t jumpStateMulti = board[jumpMulti];
														if(SQUARE_ISEMPTY(jumpStateMulti))
														{
															jumps[jumpIndex].moveType = MOVE_JUMP_MULTI;
															break;
														}
													}
												}
											}
										}
									}
								}
							}
						}
					}
				}
			}
		}
	}
	__syncthreads();

	if(jumpCount)
	{
		if(IS_ROOT_THREAD) moveCount = jumpCount;
		if(i < jumpCount) movesOut[i] = jumps[i];
	}
	else if(i < moveCount) movesOut[i] = moves[i];
	__syncthreads();
}

__device__ bool evalBoardSquareGPU(result_gpu_t* resultOut, boardstate_t* board, boardpos_t (&cornerTile)[SQUARE_COUNT][4])
{
	__shared__ int blackCount;
	__shared__ int redCount;
	__shared__ int redMoveFound;
	__shared__ int blackMoveFound;
	__shared__ bool retVal;
	if(IS_ROOT_THREAD)
	{
		blackCount = 0;
		redCount = 0;
		redMoveFound = 0;
		blackMoveFound = 0;
		retVal = false;
	}

	boardpos_t pos = threadIdx.x;
	boardstate_t state = board[pos];
	if(SQUARE_ISNOTEMPTY(state))
	{
		if(SQUARE_ISBLACK(state))
		{
			atomicAdd(&blackCount, 1);
			uint8_t cornerMin = 2;
			if(SQUARE_ISKING(state)) cornerMin = 0;
			for(uint8_t i = cornerMin; i < 4; i++)
			{
				// Get move
				boardpos_t move = cornerTile[pos][i];

				// Check if position is invalid
				if(move != BOARD_POS_INVALID)
				{
					// Check if space is empty
					boardstate_t moveState = board[move];
					if(SQUARE_ISEMPTY(moveState))
					{
						atomicAdd(&blackMoveFound, 1);
					}
					else if(!(SQUARE_ISBLACK(moveState)))
					{
						// Get jump
						boardpos_t jump = cornerTile[move][i];
						// Check if position is invalid
						if(jump != BOARD_POS_INVALID)
						{
							// Check if space is empty
							if(SQUARE_ISEMPTY(board[jump]))
							{
								// Add jump to potential moves
								atomicAdd(&blackMoveFound, 1);
							}
						}
					}
				}
			}
		}
		else
		{
			atomicAdd(&redCount, 1);
			uint8_t cornerMax = 2;
			if(SQUARE_ISKING(state)) cornerMax = 4;
			for(uint8_t i = 0; i < cornerMax; i++)
			{
				// Get move
				boardpos_t move = cornerTile[pos][i];

				// Check if position is invalid
				if(move != BOARD_POS_INVALID)
				{
					// Check if space is empty
					boardstate_t moveState = board[move];
					if(SQUARE_ISEMPTY(moveState))
					{
						atomicAdd(&redMoveFound, 1);
					}
					else if(SQUARE_ISBLACK(moveState))
					{
						// Get jump
						boardpos_t jump = cornerTile[move][i];
						// Check if position is invalid
						if(jump != BOARD_POS_INVALID)
						{
							// Check if space is empty
							if(SQUARE_ISEMPTY(board[jump]))
							{
								// Add jump to potential moves
								atomicAdd(&redMoveFound, 1);
							}
						}
					}
				}
			}
		}
	}
	__syncthreads();

	if(IS_ROOT_THREAD)
	{
		if(blackCount == 0)
		{
			if(redCount != 0)
			{
				// Red win
				*resultOut = RESULT_RED_WIN;
				retVal = true;
			}
		}
		else if(redCount == 0)
		{
			if(blackCount != 0)
			{
				// Black win
				*resultOut = RESULT_BLACK_WIN;
				retVal = true;
			}
		}
		else
		{
			if(!blackMoveFound)
			{
				if(redMoveFound)
				{
					// RED WIN
					*resultOut = RESULT_RED_WIN;
					retVal = true;
				}
				else
				{
					// TIE
					*resultOut = RESULT_TIE;
					retVal = true;
				}
			}
			else if(!redMoveFound)
			{
				if(blackMoveFound)
				{
					// BLACK WIN
					*resultOut = RESULT_BLACK_WIN;
					retVal = true;
				}
				else
				{
					// TIE
					*resultOut = RESULT_TIE;
					retVal = true;
				}
			}
		}
		if(!retVal) *resultOut = blackCount - redCount;
		if(*resultOut < -3)
		{
			//printf("Eval: %i,%i,%i,%i,%i,%i ; %i\n", *resultOut, blackCount, redCount, blackMoveFound, redMoveFound, retVal, board[0]);
		}
	}
	__syncthreads();
	return retVal;
}

// Definition
__global__ void evalRedMoveKernel(result_gpu_t* result, boardstate_t* board, Move* oldMoves, depth_t depth);

__global__ void evalBlackMoveKernel(result_gpu_t* result, boardstate_t* board, Move* oldMoves, depth_t depth)
{
	__shared__ boardpos_t cornerTile[SQUARE_COUNT][4];
	__shared__ boardstate_t boardTile[SQUARE_COUNT];
	__shared__ Move moveTile;
	__shared__ unsigned int moveCount;
	__shared__ int resultVal;
	__shared__ int resultIndex;
	result_gpu_t* resultOut = &result[blockIdx.x];
	unsigned int x = threadIdx.x;
	//if(IS_ROOT_THREAD) if(board[0] == 0 && board[1] == 0) printf("Black board empty @D%i, X:%i\n", depth, oldMoves[blockIdx.x].newPos);

	// Copy board
	boardTile[x] = board[x];

	// Copy cornerTile for faster calculations
	cornerTile[x][0] = cornerListDev[x][0];
	cornerTile[x][1] = cornerListDev[x][1];
	cornerTile[x][2] = cornerListDev[x][2];
	cornerTile[x][3] = cornerListDev[x][3];

	// Execute Move (if root)
	if(IS_ROOT_THREAD)
	{
		moveTile = oldMoves[blockIdx.x];
		boardTile[moveTile.newPos] = boardTile[moveTile.oldPos];
		boardTile[moveTile.oldPos] = SQUARE_EMPTY;
		if(MOVE_ISJUMP(moveTile)) boardTile[moveTile.jumpPos] = SQUARE_EMPTY;

		// Check for king
		if(moveTile.newPos > 27)
		{
			if(SQUARE_ISNOTEMPTY(boardTile[moveTile.newPos]))
			{
				boardTile[moveTile.newPos] |= 0x1;
			}
		}
		moveCount = 0;
		resultIndex = -1;
	}
	__syncthreads();

	// Check depth
	if(evalBoardSquareGPU(resultOut, boardTile, cornerTile))
	{
		if(IS_ROOT_THREAD)
		{
			//if(*resultOut < -3) printf("Black result @D%i: %i\n", depth, *resultOut);
		}
		return;
	}
	//if(IS_ROOT_THREAD) printf("Result @ D%i: %i\n", depth, *result);
	if(depth == NODE_DEPTH_GPU)
	{
		if(IS_ROOT_THREAD)
		{
			//printf("Red result @D%i: %i\n", depth, *resultOut);
		}
		return;
	}

	__shared__ Move* moves;
	__shared__ result_gpu_t* results;
	__shared__ boardstate_t* newBoard;
	if(IS_ROOT_THREAD)
	{
		gpuErrchk(hipMalloc(&moves, MOVE_BUFFER_SIZE*sizeof(Move)));
		gpuErrchk(hipMalloc(&newBoard, SQUARE_COUNT*sizeof(boardstate_t)));
	}
	__syncthreads();
	newBoard[x] = boardTile[x];
	__syncthreads();

	if(moveTile.moveType == MOVE_JUMP_MULTI)
	{
		// Create moves
		getBlackJumpsGPU(moves, moveCount, moveTile.newPos, boardTile, cornerTile);
		if(moveCount == 0)
		{
			getBlackMovesGPU(moves, moveCount, boardTile, cornerTile);
		}

		// Evaluate Moves (recursive)
		__syncthreads();
		/*if(moveCount == 0)
		{
			if(IS_ROOT_THREAD) //printf("Black result1 @D%i: %i ; %i\n", depth, *resultOut, boardTile[0]);
			return;
		}*/
		if(IS_ROOT_THREAD)
		{
			gpuErrchk(hipMalloc(&results, moveCount*sizeof(result_gpu_t)));
			//if(depth == 3) if(newBoard[0] == 0 && newBoard[1] == 0) printf("Null found\n");
			evalBlackMoveKernel CUDA_KERNEL(moveCount, SQUARE_COUNT) (results, newBoard, moves, depth + 1);
			gpuErrchk( hipPeekAtLastError() );
			gpuErrchk( hipDeviceSynchronize() );
			hipFree(moves);
			hipFree(newBoard);
			resultVal = RESULT_RED_WIN;
		}
		__syncthreads();

		// Pick max result
		if(threadIdx.x < moveCount)
		{
			atomicMax(&resultVal, results[threadIdx.x]);
			__syncthreads();
			if(resultVal == results[threadIdx.x])
			{
				resultIndex = x;
			}
		}
	}
	else
	{
		// Create moves
		getRedMovesGPU(moves, moveCount, boardTile, cornerTile);

		// Evaluate Moves (recursive)
		//printf("Possible Move: %i,%i,%i,%i\n", moves[x].oldPos, moves[x].newPos, moves[x].jumpPos, moves[x].moveType);
		__syncthreads();
		/*if(moveCount == 0)
		{
			if(IS_ROOT_THREAD) //printf("Black result2 @D%i: %i ; %i,%i,%i\n", depth, *resultOut, board[0], boardTile[0], newBoard[0]);
			return;
		}*/
		if(IS_ROOT_THREAD)
		{
			gpuErrchk(hipMalloc(&results, moveCount*sizeof(result_gpu_t)));
			//if(newBoard == nullptr) printf("Null found\n");
			evalRedMoveKernel CUDA_KERNEL(moveCount, SQUARE_COUNT) (results, newBoard, moves, depth + 1);
			gpuErrchk( hipPeekAtLastError() );
			gpuErrchk( hipDeviceSynchronize() );
			hipFree(moves);
			hipFree(newBoard);
			resultVal = RESULT_BLACK_WIN;
		}
		__syncthreads();

		// Pick min result
		if(threadIdx.x < moveCount)
		{
			//printf("Result from Red: %i\n", results[threadIdx.x]);
			atomicMin(&resultVal, results[threadIdx.x]);
			__syncthreads();
			if(resultVal == results[threadIdx.x])
			{
				resultIndex = x;
			}
		}
	}
	__syncthreads();
	if(IS_ROOT_THREAD)
	{
		*resultOut = results[resultIndex];
		hipFree(results);
		//printf("Black Kernel Finished @D%i\n", depth);
	}
}

__global__ void evalRedMoveKernel(result_gpu_t* result, boardstate_t* board, Move* oldMoves, depth_t depth)
{
	__shared__ boardpos_t cornerTile[SQUARE_COUNT][4];
	__shared__ boardstate_t boardTile[SQUARE_COUNT];
	__shared__ Move moveTile;
	__shared__ unsigned int moveCount;
	__shared__ int resultVal;
	__shared__ int resultIndex;
	result_gpu_t* resultOut = &result[blockIdx.x];
	unsigned int x = threadIdx.x;
	//if(threadIdx.x == 1) if(board[0] == 0 && board[1] == 0) printf("Red board empty @D%i\n", depth);

	// Copy board
	boardTile[x] = board[x];
	__syncthreads();
	//if(threadIdx.x == 1) if(boardTile[0] == 0 && boardTile[1] == 0) printf("Red boardtile empty @D%i\n", depth);

	// Copy cornerTile for faster calculations
	cornerTile[x][0] = cornerListDev[x][0];
	cornerTile[x][1] = cornerListDev[x][1];
	cornerTile[x][2] = cornerListDev[x][2];
	cornerTile[x][3] = cornerListDev[x][3];

	// Execute Move (if root)
	if(IS_ROOT_THREAD)
	{
		moveTile = oldMoves[blockIdx.x];
		boardTile[moveTile.newPos] = boardTile[moveTile.oldPos];
		boardTile[moveTile.oldPos] = SQUARE_EMPTY;
		if(MOVE_ISJUMP(moveTile)) boardTile[moveTile.jumpPos] = SQUARE_EMPTY;

		// Check for king
		if(moveTile.newPos < 4)
		{
			if(SQUARE_ISNOTEMPTY(boardTile[moveTile.newPos]))
			{
				boardTile[moveTile.newPos] |= 0x1;
			}
		}
		moveCount = 0;
		resultIndex = -1;
	}
	__syncthreads();

	// Check depth
	if(evalBoardSquareGPU(resultOut, boardTile, cornerTile))
	{
		if(IS_ROOT_THREAD)
		{
			//if(*resultOut < -3) printf("Red result @D%i: %i\n", depth, *resultOut);
		}
		return;
	}
	if(depth == NODE_DEPTH_GPU)
	{
		if(IS_ROOT_THREAD)
		{
			//printf("Red result @D%i: %i\n", depth, *resultOut);
		}
		return;
	}

	__shared__ Move* moves;
	__shared__ result_gpu_t* results;
	__shared__ boardstate_t* newBoard;
	__syncthreads();
	if(IS_ROOT_THREAD)
	{
		gpuErrchk(hipMalloc(&moves, MOVE_BUFFER_SIZE*sizeof(Move)));
		gpuErrchk(hipMalloc(&newBoard, SQUARE_COUNT*sizeof(boardstate_t)));
	}
	__syncthreads();
	newBoard[x] = boardTile[x];
	__syncthreads();
	if(moveTile.moveType == MOVE_JUMP_MULTI)
	{
		// Create moves
		getRedJumpsGPU(moves, moveCount, moveTile.newPos, boardTile, cornerTile);
		if(moveCount == 0)
		{
			getRedMovesGPU(moves, moveCount, boardTile, cornerTile);
		}

		// Evaluate Moves (recursive)
		__syncthreads();
		/*if(moveCount == 0)
		{
			if(IS_ROOT_THREAD) printf("Red result @D%i: %i ; %i\n", depth, *resultOut, boardTile[0]);
			return;
		}*/
		if(IS_ROOT_THREAD)
		{
			gpuErrchk(hipMalloc(&results, moveCount*sizeof(result_gpu_t)));
			//if(newBoard == nullptr) printf("Null found\n");
			evalRedMoveKernel CUDA_KERNEL(moveCount, SQUARE_COUNT) (results, newBoard, moves, depth + 1);
			gpuErrchk( hipPeekAtLastError() );
			gpuErrchk( hipDeviceSynchronize() );
			hipFree(moves);
			hipFree(newBoard);
			resultVal = RESULT_BLACK_WIN;
		}
		__syncthreads();

		// Pick max result
		if(threadIdx.x < moveCount)
		{
			atomicMin(&resultVal, results[threadIdx.x]);
			__syncthreads();
			if(resultVal == results[threadIdx.x])
			{
				resultIndex = x;
			}
		}
	}
	else
	{
		// Create moves
		getBlackMovesGPU(moves, moveCount, boardTile, cornerTile);
		//if(threadIdx.x == 1) if(newBoard[0] == 0 && newBoard[1] == 0) printf("Red newboard empty @D%i\n", depth);

		// Evaluate Moves (recursive)
		__syncthreads();
		/*if(moveCount == 0)
		{
			//if(IS_ROOT_THREAD) printf("Red result @D%i: %i\n", depth, *resultOut);
			return;
		}*/
		if(IS_ROOT_THREAD)
		{
			gpuErrchk(hipMalloc(&results, moveCount*sizeof(result_gpu_t)));
			//if(depth == 3) if(newBoard[0] == 0 && newBoard[1] == 0) printf("Null found2\n");
			evalBlackMoveKernel CUDA_KERNEL(moveCount, SQUARE_COUNT) (results, newBoard, moves, depth + 1);
			gpuErrchk( hipPeekAtLastError() );
			gpuErrchk( hipDeviceSynchronize() );
			hipFree(moves);
			hipFree(newBoard);
			resultVal = RESULT_RED_WIN;
		}
		__syncthreads();

		// Pick min result
		if(threadIdx.x < moveCount)
		{
			atomicMax(&resultVal, results[threadIdx.x]);
			__syncthreads();
			if(resultVal == results[threadIdx.x])
			{
				resultIndex = x;
			}
		}
	}
	__syncthreads();
	if(IS_ROOT_THREAD)
	{
		*resultOut = results[resultIndex];
		hipFree(results);
		//printf("Red Kernel Finished @D%i\n", depth);
	}
}

__global__ void getMoveKernel(Move* move, boardstate_t* board)
{
	__shared__ Move moveTile;
	__shared__ boardpos_t cornerTile[SQUARE_COUNT][4];
	__shared__ int maxResult;
	__shared__ uint16_t maxIndex;
	__shared__ unsigned int moveCount;
	__shared__ Move* moves;
	if(IS_ROOT_THREAD)
	{
		moveCount = 0;
		hipMalloc(&moves, MOVE_BUFFER_SIZE*sizeof(Move));
	}
	__syncthreads();

	unsigned int x = threadIdx.x;

	// Copy cornerTile for faster calculations
	cornerTile[x][0] = cornerListDev[x][0];
	cornerTile[x][1] = cornerListDev[x][1];
	cornerTile[x][2] = cornerListDev[x][2];
	cornerTile[x][3] = cornerListDev[x][3];
	__syncthreads();

	if(previousMultiJumpPosGPU == BOARD_POS_INVALID)
	{
		getBlackMovesGPU(moves, moveCount, board, cornerTile);
	}
	else
	{
		getBlackJumpsGPU(moves, moveCount, previousMultiJumpPosGPU, board, cornerTile);
		if(moveCount == 0)
		{
			getBlackMovesGPU(moves, moveCount, board, cornerTile);
		}
	}

	if(moveCount == 0)
	{
		if(IS_ROOT_THREAD)
		{
			moveTile.moveType = MOVE_INVALID;
			*move = moveTile;
		}
		return;
	}

	__shared__ result_gpu_t* results;
	if(IS_ROOT_THREAD)
	{
		//moveCount = 1;
		hipMalloc(&results, moveCount*sizeof(result_gpu_t));
		//printf("Possible move count: %i\n", moveCount);
		evalBlackMoveKernel CUDA_KERNEL(moveCount, SQUARE_COUNT) (results, board, moves, 0);
		gpuErrchk( hipPeekAtLastError() );
		gpuErrchk( hipDeviceSynchronize() );
		//printf("Child Kernels finished\n");
		maxResult = RESULT_RED_WIN;
		maxIndex = 0;
	}
	__syncthreads();

	// Pick result
	if(x < moveCount)
	{
		//printf("Possible Move: %i,%i,%i,%i with a result of %i\n", moves[x].oldPos, moves[x].newPos, moves[x].jumpPos, moves[x].moveType, results[x]);
		atomicMax(&maxResult, results[x]);
		__syncthreads();
		if(maxResult == results[x])
		{
			maxIndex = x;
		}
		__syncthreads();
	}
	__syncthreads();
	if(IS_ROOT_THREAD)
	{
		moveTile = moves[maxIndex];

		// Check for multijump
		if(moveTile.moveType == MOVE_JUMP_MULTI)
		{
			previousMultiJumpPosGPU = moveTile.newPos;
		}
		else previousMultiJumpPosGPU = -1;
		*move = moveTile;
		//printf("Selected Move: %i,%i,%i,%i with a result of %i\n", moveTile.oldPos, moveTile.newPos, moveTile.jumpPos, moveTile.moveType, maxResult);
	}
	__syncthreads();
}

void GPUUtility::initializeGPU()
{
	hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, NODE_DEPTH_GPU + 1);
	//hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, 2048);
	//hipDeviceSetLimit(hipLimitMallocHeapSize, 67108864);
	//hipDeviceSetLimit(hipLimitStackSize, 1024);
	size_t stack_limit;
	hipDeviceGetLimit(&stack_limit,hipLimitStackSize);
	printf("Stack size: %llu\n", stack_limit);
	size_t heap_limit;
	hipDeviceGetLimit(&heap_limit,hipLimitMallocHeapSize);
	printf("Heap size: %llu\n", heap_limit);
}

Move GPUUtility::getMove(BoardState* board)
{	
	Move *move_host, *move_dev;
	move_host = new Move;
	hipMalloc(&move_dev, sizeof(Move));

	boardstate_t *board_dev;
	hipMalloc(&board_dev, sizeof(BoardState));
	hipMemcpy(board_dev, board, sizeof(BoardState), hipMemcpyHostToDevice);

	getMoveKernel CUDA_KERNEL(1,32) (move_dev, board_dev);
	hipDeviceSynchronize();
	hipMemcpy(move_host, move_dev, sizeof(Move), hipMemcpyDeviceToHost);
	return *move_host;

	// You forgot to free memory dummy
}
