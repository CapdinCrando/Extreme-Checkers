#include <iostream>
#include <math.h>

#include <hip/hip_runtime.h>


// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
	y[i] = x[i] + y[i];
}
